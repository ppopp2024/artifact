#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <complex>
#include "kernels.cuh"
#include <hip/hip_runtime.h> 
#include <hipfft/hipfftXt.h>
#include "utils/utils.cuh"   
#define FLOAT2_NORM(a, res) res = a.x * a.x + a.y * a.y;
int main(int argc, char** argv){  
    // #if (V == 1)
    int __log_N__, __log_N_st__ = 3;
    float * t_cufft, *t_vkfft, *t_fft;
    t_cufft = (float*)malloc(sizeof(float) * 34);
    t_vkfft = (float*)malloc(sizeof(float) * 34);
    t_fft = (float*)malloc(sizeof(float) * 34);
    
    if (argc < 2){
        printf("Please input log(N)\n");
        return -1;
    }
    else if(argc == 2) __log_N__ = atoi(argv[1]);
    else if(argc == 3){
        __log_N__ = atoi(argv[2]);
        __log_N_st__ = atoi(argv[1]);
    }
    // #endif
    int N = pow((double)2, (double)__log_N__); 
    int random_seed = 10;  
    #if P_FFT == 1
    int num_tests = 100;
    #else
    int num_tests = 1;
    #endif
    srandom(random_seed); 
    float *input = (float*)calloc(N * 2, sizeof(float)); 
    float *output_ref, *output;
    
    output_ref = (float*)calloc(N * 2, sizeof(float));
    output = (float*)calloc(N * 2, sizeof(float));
    
    float *input_d, *output_d, *output_d_vkfft, *output_d_cufft, *output_d_1, *output_d_ref_1;
 
    CUDA_CALLER(hipMalloc((void**)&input_d, sizeof(float) * N * 2));
    CUDA_CALLER(hipMalloc((void**)&output_d, sizeof(float) * N * 2));
    // CUDA_CALLER(hipMalloc((void**)&output_d_vkfft, sizeof(float) * N * 2));
    // CUDA_CALLER(hipMalloc((void**)&output_d_cufft, sizeof(float) * N * 2));
    CUDA_CALLER(hipMalloc((void**)&output_d_1, sizeof(float) * N * 2));
    // CUDA_CALLER(hipMalloc((void**)&output_d_ref_1, sizeof(float) * N * 2));

    for(int i = 0; i < N * 2; ++i){ 
            input[i] = (float)(random() % 100) / (float)100;   
    }
    hipMemcpy((void*)input_d, (void*)input, 2 * N * sizeof(float), hipMemcpyHostToDevice);

    hipfftHandle plan;  
    hipfftCreate(&plan);


    hipEvent_t fft_begin, fft_end;
    float elapsed_time_vkfft, elapsed_time, elapsed_time_cufft; 
    std::chrono::steady_clock::time_point timeSt; // = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point timeEnd; // = std::chrono::steady_clock::now();
	float totTime, totTime_vkfft, totTime_cufft;
    hipEventCreate(&fft_begin);
    hipEventCreate(&fft_end);
    
    #if P_FFT == 1
    for(int log_N = __log_N_st__; log_N <= __log_N__; ++log_N){
    #else
    int log_N = __log_N__;
    #endif
    N = pow(double(2), double(log_N));
    if(log_N == 3){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(1, 1, 1);
            // for(int i = 0; i < num_tests; ++i){
            //     fft_radix2_logN3 <<<gridDim, blockDim, 64>>> ((float2*)input_d, (float2*)output_d);
            //     hipDeviceSynchronize();
            // }
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN3 <<<gridDim, blockDim, 192>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(1, 1, 1);
            // for(int i = 0; i < num_tests; ++i){
            //     VkFFT_main_logN3 <<<gridDim, blockDim, 192>>>((float2*)input_d, (float2*)output_d_vkfft);
            //     hipDeviceSynchronize();  
            // }
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN3 <<<gridDim, blockDim, 192>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 4){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(4, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN4 <<<gridDim, blockDim, 256>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN4 <<<gridDim, blockDim, 256>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(2, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN4 <<<gridDim, blockDim, 256>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN4 <<<gridDim, blockDim, 256>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 5){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(4, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN5 <<<gridDim, blockDim, 384>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN5 <<<gridDim, blockDim, 384>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(4, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN5 <<<gridDim, blockDim, 384>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN5 <<<gridDim, blockDim, 384>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 6){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(8, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN6 <<<gridDim, blockDim, 640>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN6 <<<gridDim, blockDim, 640>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(8, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN6 <<<gridDim, blockDim, 640>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN6 <<<gridDim, blockDim, 640>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 7){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(16, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN7 <<<gridDim, blockDim, 1152>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN7 <<<gridDim, blockDim, 1152>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(16, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN7 <<<gridDim, blockDim, 1152>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN7 <<<gridDim, blockDim, 1152>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 8){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(32, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN8 <<<gridDim, blockDim, 2176>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN8 <<<gridDim, blockDim, 2176>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(32, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN8 <<<gridDim, blockDim, 2176>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN8 <<<gridDim, blockDim, 2176>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 9){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(64, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN9 <<<gridDim, blockDim, 4352>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN9 <<<gridDim, blockDim, 4352>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(64, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN9 <<<gridDim, blockDim, 4352>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN9 <<<gridDim, blockDim, 4352>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 10){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(128, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN10 <<<gridDim, blockDim, 8704>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN10 <<<gridDim, blockDim, 8704>>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(128, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN10 <<<gridDim, blockDim, 8704>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN10 <<<gridDim, blockDim, 8704>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 11){
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(128, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN11 <<<gridDim, blockDim, 17408 >>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN11 <<<gridDim, blockDim, 17408 >>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
                {
            // dim3 gridDim(1, 1, 1);
            // dim3 blockDim(256, 1, 1);
            // for(int i = 0; i < num_tests; ++i){
            //     fft_radix2_logN11 <<<gridDim, blockDim, 16384>>> ((float2*)input_d, (float2*)output_d_1);
            //     hipDeviceSynchronize();
            // }
            // hipEventRecord(fft_begin);
            // for(int i = 0; i < num_tests; ++i){
            //     fft_radix2_logN11 <<<gridDim, blockDim, 16384>>> ((float2*)input_d, (float2*)output_d_1);
            //     hipDeviceSynchronize();
            // }    
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(128, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN11 <<<gridDim, blockDim, 17408>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN11 <<<gridDim, blockDim, 17408>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
    }
    else if(log_N == 12){
        // {
        //     dim3 gridDim(1, 1, 1);
        //     dim3 blockDim(512, 1, 1);
        //     for(int i = 0; i < num_tests; ++i){
        //         fft_radix2_logN12 <<<gridDim, blockDim, 32768>>> ((float2*)input_d, (float2*)output_d_1);
        //         hipDeviceSynchronize();
        //     }
        //     hipEventRecord(fft_begin);
        //     for(int i = 0; i < num_tests; ++i){
        //         fft_radix2_logN12 <<<gridDim, blockDim, 32768>>> ((float2*)input_d, (float2*)output_d_1);
        //         hipDeviceSynchronize();
        //     }    
        //     hipEventRecord(fft_end);  
        //     hipEventSynchronize(fft_begin);
        //     hipEventSynchronize(fft_end);
        //     hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        // }
        
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(256, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN12 <<<gridDim, blockDim, 34816 >>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN12 <<<gridDim, blockDim, 34816 >>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(256, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN12 <<<gridDim, blockDim, 34816>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN12 <<<gridDim, blockDim, 34816>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
    }
    else if(log_N == 13){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN13), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN13), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(1024, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN13 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                VkFFT_main_logN13 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_vkfft);
                hipDeviceSynchronize();  
            }
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            dim3 gridDim(1, 1, 1);
            dim3 blockDim(1024, 1, 1);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN13 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                fft_radix2_logN13 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d);
                hipDeviceSynchronize();
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);    
        }
    }
    else if(log_N == 14){
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(16, 1, 1);
                dim3 blockDim(16, 8, 1);
                fft_radix2_logN14_1 <<<gridDim, blockDim, 8192>>> ((float2*)input_d, (float2*)output_d_1);       
                // hipDeviceSynchronize();
                {
                    dim3 gridDim(4, 1, 1);
                    dim3 blockDim(16, 32, 1); 
                    fft_radix2_logN14_2 <<<gridDim, blockDim, 32768>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(16, 1, 1);
                dim3 blockDim(16, 8, 1);
                VkFFT_main_logN14_1 <<<gridDim, blockDim, 8192 >>>((float2*)input_d, (float2*)output_d_ref_1);
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(4, 1, 1);
                dim3 blockDim(16, 32, 1); 
                VkFFT_main_logN14_2 <<<gridDim, blockDim, 34816 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 15){
        
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN15_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN15_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 8, 1);
                fft_radix2_logN15_1 <<<gridDim, blockDim, 8192>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(4, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN15_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 8, 1);
                VkFFT_main_logN15_1 <<<gridDim, blockDim, 8192 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(4, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN15_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 16){
        
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN16_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN16_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 16, 1);
                fft_radix2_logN16_1 <<<gridDim, blockDim, 16384>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(8, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN16_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 16, 1);
                VkFFT_main_logN16_1 <<<gridDim, blockDim, 16384 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(8, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN16_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 17){
        {
            hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN17_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
            hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN17_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 16, 1);
                fft_radix2_logN17_1 <<<gridDim, blockDim, 32768>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(16, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN17_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 16, 1);
                VkFFT_main_logN17_1 <<<gridDim, blockDim, 32768 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(16, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN17_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 18){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN18_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN18_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN18_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN18_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 64, 1);
                fft_radix2_logN18_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN18_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {    
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(16, 64, 1);
                VkFFT_main_logN18_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(32, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN18_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
        }
    }
    else if(log_N == 19){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN19_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN19_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(64, 1, 1);
                dim3 blockDim(16, 64, 1);
                fft_radix2_logN19_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(128, 1, 1);
                dim3 blockDim(4, 32, 1); 
                fft_radix2_logN19_2 <<<gridDim, blockDim, 32768>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(64, 1, 1);
                dim3 blockDim(16, 64, 1);
                VkFFT_main_logN19_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(128, 1, 1);
                dim3 blockDim(4, 32, 1); 
                VkFFT_main_logN19_2 <<<gridDim, blockDim, 40960 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
    }
    else if(log_N == 20){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN20_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN20_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(128, 1, 1);
                dim3 blockDim(8, 32, 1);
                fft_radix2_logN20_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(256, 1, 1);
                dim3 blockDim(4, 32, 1); 
                fft_radix2_logN20_2 <<<gridDim, blockDim, 32768>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(128, 1, 1);
                dim3 blockDim(8, 32, 1);
                VkFFT_main_logN20_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(256, 1, 1);
                dim3 blockDim(4, 32, 1); 
                VkFFT_main_logN20_2 <<<gridDim, blockDim, 40960 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
    }
    else if(log_N == 21){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN21_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN21_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN21_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN21_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(256, 1, 1);
                dim3 blockDim(8, 32, 1);
                VkFFT_main_logN21_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(256, 1, 1);
                dim3 blockDim(128, 4, 1); 
                VkFFT_main_logN21_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(256, 1, 1);
                dim3 blockDim(8, 32, 1);
                fft_radix2_logN21_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(256, 1, 1);
                dim3 blockDim(128, 4, 1); 
                fft_radix2_logN21_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 22){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN22_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN22_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN22_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN22_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                
                dim3 gridDim(512, 1, 1);
                dim3 blockDim(4, 128, 1);
                VkFFT_main_logN22_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_ref_1);
                
                // hipDeviceSynchronize();  
                {
                dim3 gridDim(512, 1, 1);
                dim3 blockDim(128, 4, 1); 
                VkFFT_main_logN22_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(512, 1, 1);
                dim3 blockDim(4, 128, 1);
                fft_radix2_logN22_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d_1);
                
                // hipDeviceSynchronize();
                {
                dim3 gridDim(512, 1, 1);
                dim3 blockDim(128, 4, 1); 
                fft_radix2_logN22_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 23){
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(2048, 1, 1);
                dim3 blockDim(16, 16, 1);
                VkFFT_main_logN23_1 <<<gridDim, blockDim, 32768 >>>((float2*)input_d, (float2*)output_d_vkfft);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1); 
                VkFFT_main_logN23_2 <<<gridDim, blockDim, 16384 >>>((float2*)output_d_vkfft, (float2*)output_d_ref_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(2048, 1, 1);
                dim3 blockDim(16, 16, 1); 
                VkFFT_main_logN23_3 <<<gridDim, blockDim, 34816 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(2048, 1, 1);
                dim3 blockDim(16, 16, 1);
                fft_radix2_logN23_1 <<<gridDim, blockDim, 32768>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1); 
                fft_radix2_logN23_2 <<<gridDim, blockDim, 16384>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(2048, 1, 1);
                dim3 blockDim(16, 16, 1); 
                fft_radix2_logN23_3 <<<gridDim, blockDim, 32768>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 24){
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1);
                VkFFT_main_logN24_1 <<<gridDim, blockDim, 32768 >>>((float2*)input_d, (float2*)output_d_vkfft);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1); 
                VkFFT_main_logN24_2 <<<gridDim, blockDim, 32768 >>>((float2*)output_d_vkfft, (float2*)output_d_ref_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1); 
                VkFFT_main_logN24_3 <<<gridDim, blockDim, 34816 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1);
                fft_radix2_logN24_1 <<<gridDim, blockDim, 32768>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1); 
                fft_radix2_logN24_2 <<<gridDim, blockDim, 32768>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(16, 16, 1); 
                fft_radix2_logN24_3 <<<gridDim, blockDim, 32768>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 25){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN25_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN25_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(16, 16, 1);
                VkFFT_main_logN25_1 <<<gridDim, blockDim, 32768 >>>((float2*)input_d, (float2*)output_d_vkfft);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(16, 16, 1); 
                VkFFT_main_logN25_2 <<<gridDim, blockDim, 32768 >>>((float2*)output_d_vkfft, (float2*)output_d_ref_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN25_3 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(16, 16, 1);
                fft_radix2_logN25_1 <<<gridDim, blockDim, 32768>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(16, 16, 1); 
                fft_radix2_logN25_2 <<<gridDim, blockDim, 32768>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(4096, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN25_3 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 26){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN26_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN26_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN26_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN26_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(16, 64, 1);
                VkFFT_main_logN26_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_vkfft);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(16, 16, 1); 
                VkFFT_main_logN26_2 <<<gridDim, blockDim, 32768 >>>((float2*)output_d_vkfft, (float2*)output_d_ref_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN26_3 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(16, 64, 1);
                fft_radix2_logN26_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(16, 16, 1); 
                fft_radix2_logN26_2 <<<gridDim, blockDim, 32768>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(8192, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN26_3 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 27){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN27_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN27_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN27_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN27_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN27_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN27_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(16, 64, 1);
                VkFFT_main_logN27_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_vkfft);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(16, 64, 1); 
                VkFFT_main_logN27_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_vkfft, (float2*)output_d_ref_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(64, 16, 1); 
                VkFFT_main_logN27_3 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(16, 64, 1);
                fft_radix2_logN27_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(16, 64, 1); 
                fft_radix2_logN27_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(16384, 1, 1);
                dim3 blockDim(64, 16, 1); 
                fft_radix2_logN27_3 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    else if(log_N == 28){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN28_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN28_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN28_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN28_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN28_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN28_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(32768, 1, 1);
                dim3 blockDim(16, 64, 1);
                VkFFT_main_logN28_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d_vkfft);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(32768, 1, 1);
                dim3 blockDim(16, 64, 1); 
                VkFFT_main_logN28_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_vkfft, (float2*)output_d_ref_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(32768, 1, 1);
                dim3 blockDim(32, 8, 1); 
                VkFFT_main_logN28_3 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_ref_1, (float2*)output_d_vkfft);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d_cufft, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(32768, 1, 1);
                dim3 blockDim(16, 64, 1);
                fft_radix2_logN28_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(32768, 1, 1);
                dim3 blockDim(16, 64, 1); 
                fft_radix2_logN28_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(32768, 1, 1);
                dim3 blockDim(32, 8, 1); 
                fft_radix2_logN28_3 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
        }
    }
    if(log_N == 29){
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN29_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN29_1), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN29_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN29_2), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(fft_radix2_logN29_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        hipFuncSetAttribute(reinterpret_cast<const void*>(VkFFT_main_logN29_3), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                dim3 gridDim(65536, 1, 1);
                dim3 blockDim(8, 32, 1);
                VkFFT_main_logN29_1 <<<gridDim, blockDim, 65536 >>>((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();  
                {
                dim3 gridDim(65536, 1, 1);
                dim3 blockDim(16, 64, 1); 
                VkFFT_main_logN29_2 <<<gridDim, blockDim, 65536 >>>((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize();  
                {
                dim3 gridDim(65536, 1, 1);
                dim3 blockDim(32, 8, 1); 
                VkFFT_main_logN29_3 <<<gridDim, blockDim, 65536 >>>((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize();  
            }
            timeEnd = std::chrono::steady_clock::now();
            totTime_vkfft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);
            hipEventSynchronize(fft_begin);  
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_vkfft, fft_begin, fft_end);
            hipMemcpy((void*)output_ref, (void*)output_d, sizeof(float) * 2 * N, hipMemcpyDeviceToHost);
        }
        {
            hipfftPlan1d(&plan, N, HIPFFT_C2C, 1); 
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){
                hipfftExecC2C(plan, (hipfftComplex *)input_d, (hipfftComplex *)output_d, HIPFFT_FORWARD);
                hipDeviceSynchronize(); 
            } 
            timeEnd = std::chrono::steady_clock::now();
            totTime_cufft = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time_cufft, fft_begin, fft_end);   
            
        }
        {
            hipEventRecord(fft_begin);
            timeSt = std::chrono::steady_clock::now();
            for(int i = 0; i < num_tests; ++i){    
                dim3 gridDim(65536, 1, 1);
                dim3 blockDim(8, 32, 1);
                fft_radix2_logN29_1 <<<gridDim, blockDim, 65536>>> ((float2*)input_d, (float2*)output_d);
                
                hipDeviceSynchronize();
                {
                dim3 gridDim(65536, 1, 1);
                dim3 blockDim(16, 64, 1); 
                fft_radix2_logN29_2 <<<gridDim, blockDim, 65536>>> ((float2*)output_d, (float2*)output_d_1);
                }
                hipDeviceSynchronize(); 
                {
                dim3 gridDim(65536, 1, 1);
                dim3 blockDim(32, 8, 1); 
                fft_radix2_logN29_3 <<<gridDim, blockDim, 65536>>> ((float2*)output_d_1, (float2*)output_d);
                }
                hipDeviceSynchronize(); 
            }    
            timeEnd = std::chrono::steady_clock::now();
            totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSt).count();
            hipEventRecord(fft_end);  
            hipEventSynchronize(fft_begin);
            hipEventSynchronize(fft_end);
            hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
            hipMemcpy((void*)output, (void*)output_d, sizeof(float) * 2 * N, hipMemcpyDeviceToHost);
        }
    }
    #if V_FFT == 1
    // hipMemcpy((void*)output_ref, (void*)output_d_vkfft, 2 * N * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy((void*)output, (void*)output_d, sizeof(float) * 2 * N, hipMemcpyDeviceToHost);
    // hipMemcpy((void*)output, (void*)output_d_cufft, sizeof(float) * 2 * N, hipMemcpyDeviceToHost);
    // hipMemcpy((void*)output, (void*)output_d_ref_1, 2 * N * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy((void*)output_ref, (void*)output_d_1, sizeof(float) * 2 * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    bool pass = true;
    for(int i = 0; i < 2 * N; i +=2){
        float2 res = *(float2*)(output + i); 
        float2 res_ref = *(float2*)(output_ref + i);
        float norm, norm_ref; 
        FLOAT2_NORM(res, norm);
        FLOAT2_NORM(res_ref, norm_ref);
        
        float err = fabs(norm - norm_ref);
        if(i % 100000 ==0){
        printf("error %f detected at %d\n", err / fabs(norm), i / 2);
        printf("ref[%d]: %.3f + %.3f i\n",  i / 2, res_ref.x, res_ref.y);
        printf("res[%d]: %.3f + %.3f i\n\n",  i / 2, res.x, res.y);
        }
        if(err / fabs(norm) > 0.05){
            printf("error %f detected at %d\n", err / fabs(norm), i / 2);
            printf("ref[%d]: %.3f + %.3f i\n",  i / 2, res_ref.x, res_ref.y);
            printf("res[%d]: %.3f + %.3f i\n\n",  i / 2, res.x, res.y);
            pass = false;
            break;
        }   
    }
    if(pass) printf("Pass!\n");
    else printf("Fail!\n");
    #endif

    #if P_FFT == 1
    elapsed_time /= num_tests;
    elapsed_time_vkfft /= num_tests;
    elapsed_time_cufft /= num_tests;
    totTime /= num_tests;
    totTime_vkfft /= num_tests;
    totTime_cufft /= num_tests;
    if(log_N == __log_N_st__)printf("| SIZE |  Execution Time (us)             |   Shared   | #threads |\n");
    if(log_N == __log_N_st__)printf("|log(N)|   Ours   |   VkFFT   |   cuFFT   | Memory (KB)|          |\n");
    printf("|%6d| %8.3f | %8.3f  |%8.3f   |%8.3f    |%10d|\n", int(log2f((float)N)), elapsed_time * 1000, elapsed_time_vkfft * 1000, elapsed_time_cufft * 1000, (float)sizeof(float) * (float)N * 2.f / 1024.f, N / 8);
    t_fft[log_N] = elapsed_time;
    t_cufft[log_N] = elapsed_time_cufft;
    t_vkfft[log_N] = elapsed_time_vkfft;
    }
    printf("Execution Time\n");
    printf("t_fft = th.as_tensor([");
    for(int i = 3; i <= __log_N__; ++i ){
        printf("%8f,", t_fft[i]);
    }
    printf("])\n");

    printf("t_cufft = th.as_tensor([");
    for(int i = 3; i <= __log_N__; ++i ){
        printf("%8f,", t_cufft[i]);
    }
    printf("])\n");
    
    printf("t_vkfft = th.as_tensor([");
    for(int i = 3; i <= __log_N__; ++i ){
        printf("%8f,", t_vkfft[i]);
    }
    printf("])\n");

    printf("\n Flops\n");
    printf("gflops_fft = th.as_tensor([");
    for(int i = 3; i <= __log_N__; ++i ){
        int N = pow((double)2, (double)i);
        printf("%8f,", 5 * N * i / t_fft[i] * 1000.f / 1000000000.f);
    }
    printf("])\n");

    printf("gflops_cufft = th.as_tensor([");
    for(int i = 3; i <= __log_N__; ++i ){
        int N = pow((double)2, (double)i);
        printf("%8f,", 5 * N * i / t_cufft[i] * 1000.f / 1000000000.f);
    }
    printf("])\n");
    
    printf("gflops_vkfft = th.as_tensor([");
    for(int i = 3; i <= __log_N__; ++i ){
        int N = pow((double)2, (double)i);
        printf("%8f,", 5 * N * i / t_vkfft[i] * 1000.f / 1000000000.f);
    }
    printf("])\n");
    #endif
    return 0;
}
