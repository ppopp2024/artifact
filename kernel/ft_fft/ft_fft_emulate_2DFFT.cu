#include "hip/hip_runtime.h"
#include <complex>
#include <iostream> 
#include <random>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <math.h> 
#include "utils/utils.cuh"          
#include "kernels.cuh"  

#define M_PI 3.14159265358979312f 
// #define VERIFY 0
#define hadmard 1
struct M_complex{
    float x, y;
};
#define M_complex_MUL(a, b, c) c.x = a.x * b.x - a.y * b.y; c.y = a.y * b.x + a.x * b.y;
void generate_W(float* W, int N){  
    for(int c = 0; c < N; ++c)
    for(int r = 0; r < N; ++r){d
        float angle = (-2.f * M_PI * r * c) / N;
        W[2 * (r + c * N)] = cosf(angle);
        W[2 * (r + c * N) + 1] = sinf(angle);
        // printf("%f + %f i\n", cosf(angle), sinf(angle));
    }
}
int main(int argc, char *argv[]) {
    if(argc != 2){
        printf("Missing exponent of FFT size as input. Default base is 2.\n");
        return -1;
    }
    // printf("%f\n", -2 * M_PI * ((1 * 2) / 2) * ((1 * 2) % 4) / (100));
    int size = atoi(argv[1]);
    int batch_size = 1;
    int num_tests = 3;
    double error_bound = 0.1;  
    int M = pow((double)2, (double)(size / 2));
    float *X, *W, *Y, *Y_ref;
    float *d_X0, *d_X1, *d_W, *d_Y, *d_Y_ref;
    float elapsed = 0, elapsed_cuFFT = 0;
    M_complex alpha, beta;
    alpha.x = 1;
    alpha.y = 0;
    beta.x = 0;
    beta.y = 0;
    X = (float*)malloc(sizeof(float) * M * M * 2);
    Y = (float*)malloc(sizeof(float) * M * M * 2);
    Y_ref = (float*)malloc(sizeof(float) * M * M * 2);
    W = (float*)malloc(sizeof(float) * M * M * 2);
    hipMalloc((void**)(&d_X1), sizeof(float) * M * M * 2);
    hipMalloc((void**)(&d_X0), sizeof(float) * M * M * 2);
    hipMalloc((void**)(&d_Y), sizeof(float) * M * M * 2);
    hipMalloc((void**)(&d_Y_ref), sizeof(float) * M * M * 2);
    hipMalloc((void**)(&d_W), sizeof(float) * M * M * 2);

    // Random init X
    generate_random_matrix(X, M);
    generate_random_matrix(X + M * M, M);
    generate_W(W, M);
    

    hipMemcpy(d_X0, X, sizeof(float) * M * M * 2, hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, sizeof(float) * M * M * 2, hipMemcpyHostToDevice);


    hipblasHandle_t handle;
    hipfftHandle fft_handle, fft_handle_test, fft_handle_test2;
    hipblasCreate(&handle); 
    hipfftCreate(&fft_handle);
    hipfftCreate(&fft_handle_test);
    hipfftCreate(&fft_handle_test2);
    dim3 blockDim(256);
    dim3 gridDim(CEIL_DIV(M * M, 512));
    
    
    hipEvent_t beg, end, beg1, end1; 
    hipEventCreate(&beg);                              
    hipEventCreate(&end);   
    hipEventCreate(&beg1);                        
    hipEventCreate(&end1);  

    /////////////////////////////////////////////////////////////////////////////////////
    // X_j x W = X_s
    // hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    //             M, M, M, 
    //             (hipComplex*)&alpha, (hipComplex*)d_X0, M, (hipComplex*)d_W, M, 
    //             (hipComplex*)&beta, (hipComplex*)d_X1, M);
    
    // hipfftPlan1d(&fft_handle_test, M * M, HIPFFT_C2C, M);
    // hipfftExecC2C(fft_handle_test, (hipComplex*)d_X0, (hipComplex*)d_X1, HIPFFT_FORWARD);
    // // Hadamard(X_s, W_v) --> X_s
    // #ifdef hadmard
    // my_hadamard_kernel<<<gridDim, blockDim>>>(d_X1, M);
    // #endif
    // // W x X_s = Y
    // hipblasCgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
    //             M, M, M, 
    //             (hipComplex*)&alpha, (hipComplex*)d_X1, M, (hipComplex*)d_W, M, 
    //             (hipComplex*)&beta, (hipComplex*)d_Y, M);
    /////////////////////////////////////////////////////////////////////////////////////
    // hipfftPlan1d(&fft_handle_test, M * M, HIPFFT_C2C, M);
    int s = M * M;
    hipfftPlanMany(&fft_handle_test, 1, &M, &M, M, 1, &M, M, 1,HIPFFT_C2C,M);
    hipfftPlanMany(&fft_handle_test2, 1, &M, &M, 1, M, &M, M, 1,HIPFFT_C2C,M);



    hipfftExecC2C(fft_handle_test, (hipComplex*)d_X0, (hipComplex*)d_X1, HIPFFT_FORWARD);
    // Hadamard(X_s, W_v) --> X_s
    #ifdef hadmard
    my_hadamard_kernel<<<gridDim, blockDim>>>(d_X1, M);
    #endif
    // W x X_s = Y
    // hipblasCgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
    //             M, M, M, 
    //             (hipComplex*)&alpha, (hipComplex*)d_X1, M, (hipComplex*)d_W, M, 
    //             (hipComplex*)&beta, (hipComplex*)d_Y, M);
    hipfftExecC2C(fft_handle_test2, (hipComplex*)d_X1, (hipComplex*)d_Y, HIPFFT_FORWARD);
    /////////////////////////////////////////////////////////////////////////////////////
    
    #ifdef VERIFY
    hipMemcpy(Y, d_Y, sizeof(float) * M * M * 2, hipMemcpyDeviceToHost);
    #endif
    
    
    // compare with cuFFT.
    hipfftPlan1d(&fft_handle, M * M, HIPFFT_C2C, batch_size);
    
    hipfftExecC2C(fft_handle, (hipComplex*)d_X0, (hipComplex*)d_Y_ref, HIPFFT_FORWARD);

    // #ifdef VERIFY
    // hipMemcpy(Y_ref, d_Y_ref, sizeof(float) * M * M * 2, hipMemcpyDeviceToHost);
    
    // compare
    // bool if_verified = true; 
    // for(int i = 0; i < M * M * 2; ++i){
    //     if(fabs(Y[i]- Y_ref[i]) / sqrt(Y_ref[(i / 2) * 2] * Y_ref[(i / 2) * 2] + Y_ref[(i / 2) * 2 + 1] * Y_ref[(i / 2) * 2 + 1] + 0.0001) > error_bound ){
    //         if_verified=false; 
    //         printf("Detect error at %d, cuFFT result: %.3f + %.3f i, our result: %.3f + %.3f i \n", 
    //         i / 2, Y_ref[(i / 2) * 2], Y_ref[(i / 2) * 2 + 1], Y[(i / 2) * 2], Y[(i / 2) * 2 + 1]);
    //     }
    // }
    // if(if_verified)printf("No error detected!\n");
    // #endif
    hipEventRecord(beg1);  
    // hipfftPlanMany(&fft_handle_test, 1, &M, &M, M, 1, &M, M, 1,HIPFFT_C2C,M);
    for(int i = 0; i < num_tests; ++i){
        
        // hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        //         M, M, M, 
        //         (hipComplex*)&alpha, (hipComplex*)d_X0, M, (hipComplex*)d_W, M, 
        //         (hipComplex*)&beta, (hipComplex*)d_X1, M);

        
        hipfftExecC2C(fft_handle_test, (hipComplex*)d_X0, (hipComplex*)d_X1, HIPFFT_FORWARD);

        // Hadamard(X_s, W_v) --> X_s
        // #ifdef hadmard
        // my_hadamard_kernel<<<gridDim, blockDim>>>(d_X1, M);
        // #endif
        // W x X_s = Y
        // hipblasCgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
        //             M, M, M, 
        //             (hipComplex*)&alpha, (hipComplex*)d_X1, M, (hipComplex*)d_W, M, 
        //             (hipComplex*)&beta, (hipComplex*)d_Y, M);
        hipfftExecC2C(fft_handle_test2, (hipComplex*)d_X1, (hipComplex*)d_Y, HIPFFT_FORWARD);
        hipDeviceSynchronize(); 
    }
    hipEventRecord(end1);     
    hipEventSynchronize(beg1);
    hipEventSynchronize(end1); 
    hipEventElapsedTime(&elapsed, beg1, end1);     


    hipEventRecord(beg);  
    for(int i = 0; i < num_tests; ++i){
        hipfftExecC2C(fft_handle, (hipComplex*)d_X0, (hipComplex*)d_Y_ref, HIPFFT_FORWARD);
        hipDeviceSynchronize(); 
    }
    hipEventRecord(end);     
    hipEventSynchronize(beg);
    hipEventSynchronize(end); 
    hipEventElapsedTime(&elapsed_cuFFT, beg, end);     

    double gflops = 0.;
    gflops = double(num_tests * 5.0 * double(M * M) * double(log2f(M * M)) ) / (1000000000.0);
    double perf =  (elapsed / num_tests) * 1000;
    double perf_cuFFT =  (elapsed_cuFFT / num_tests) * 1000;

    printf("2^%d, %.3f, %.3f\n", size, perf, perf_cuFFT);


    hipFree(d_X0);
    hipFree(d_X1);
    hipFree(d_Y);
    hipFree(d_W);
    hipFree(d_Y_ref);

    free(X);
    free(Y);
    free(W);
    free(Y_ref);


}
